void captureGpuErrors(int *gpuErrorFlag){

    int hostErrorFlag = 0;
    const int divByZero = -1;
    hipError_t err = hipGetLastError();

    // Get the last CUDA error state
    err = hipGetLastError();
    if (err != hipSuccess) throw CudaException(err);

    if (gpuErrorFlag){
        // Copy the CUDA kernel error state (things we raised in our own programming)
        err = hipMemcpy(&hostErrorFlag, gpuErrorFlag, sizeof(int), hipMemcpyDeviceToHost);

        if (err != hipSuccess) throw CudaException(err);
        if (hostErrorFlag == divByZero) throw DivisionByZeroException();
    }
}