// matrix_add.cu
#include <hip/hip_runtime.h>
#include <iostream>
#include "exceptions.h"
#include "kernels/checkZeroKernel.h"
#include "kernels/add.h"
#include "kernels/divide.h"
#include "kernels/multiply.h"

#define CUDA_FREE_ALL(a,b,c) \
    hipFree(a); \
    hipFree(b); \
    hipFree(c);
/*
 * CUDA interface function: Add two Matrices
 *
 *      c = a + b, return error_code (-1) or success (0)
 */
extern "C" int matrix_add(double* a, double* b, double* c, int rows, int cols) {
    int size = rows * cols;
    double* gpu_a = nullptr;
    double* gpu_b = nullptr;
    double* gpu_c = nullptr;

    try {
        hipError_t err;
        // Allocate device memory
        err = hipMalloc((void**)&gpu_a, size * sizeof(double));
        if (err != hipSuccess) throw CudaMallocExceptionGpuA();

        err = hipMalloc((void**)&gpu_b, size * sizeof(double));
        if (err != hipSuccess) throw CudaMallocExceptionGpuB();

        err = hipMalloc((void**)&gpu_c, size * sizeof(double));
        if (err != hipSuccess) throw CudaMallocExceptionGpuC();

        // Copy data to device
        err = hipMemcpy(gpu_a, a, size * sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) throw CudaMemcpyExceptionA();

        err = hipMemcpy(gpu_b, b, size * sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) throw CudaMemcpyExceptionB();

        // Launch kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        matrixAddKernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_a, gpu_b, gpu_c, size);

        err = hipGetLastError();
        if (err != hipSuccess) throw KernelLaunchException();

        // Copy result back to host
        err = hipMemcpy(c, gpu_c, size * sizeof(double), hipMemcpyDeviceToHost);
        if (err != hipSuccess) throw CudaMemcpyExceptionC();

    } catch (const CudaMallocExceptionGpuA& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -1;
    } catch (const CudaMallocExceptionGpuB& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -2;
    } catch (const CudaMallocExceptionGpuC& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -3;
    } catch (const CudaMallocExceptionErrorFlag& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -4;
    } catch (const CudaMemcpyExceptionA& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -5;
    } catch (const CudaMemcpyExceptionB& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -6;
    } catch (const CudaMemcpyExceptionC& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -7;
    } catch (const KernelLaunchException& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -8;
    } catch (const DivisionByZeroException& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -9;
    } catch (const std::runtime_error& e) {
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
         return -1;
     }
    CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
    return 0;  // Return success code
}

/*
 * CUDA interface function: Divide two Matrices
 *
 *      c = a / b, return error_code (-1) or success (0)
 */
extern "C" int matrix_divide(double* a, double* b, double* c, int rows, int cols) {
    int size = rows * cols;
    double* gpu_a = nullptr;
    double* gpu_b = nullptr;
    double* gpu_c = nullptr;

    try {
        int* d_errorFlag = 0;
        hipError_t err;
        // Allocate device memory
        err = hipMalloc((void**)&gpu_a, size * sizeof(double));
        if (err != hipSuccess) throw CudaMallocExceptionGpuA();

        err = hipMalloc((void**)&gpu_b, size * sizeof(double));
        if (err != hipSuccess) throw CudaMallocExceptionGpuB();

        err = hipMalloc((void**)&gpu_c, size * sizeof(double));
        if (err != hipSuccess) throw CudaMallocExceptionGpuC();

        err = hipMalloc((void**)&d_errorFlag, sizeof(int));
        if (err != hipSuccess) throw CudaMallocExceptionErrorFlag();

        // Copy data to device
        err = hipMemcpy(gpu_a, a, size * sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) throw CudaMemcpyExceptionA();

        err = hipMemcpy(gpu_b, b, size * sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) throw CudaMemcpyExceptionB();

        int h_errorFlag = 0;
        err = hipMemcpy(d_errorFlag, &h_errorFlag, sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) throw CudaMemcpyExceptionErrorFlag();

        // Check for zero values in divisor
        int threadsPerBlock = 256;
        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        checkZeroKernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_b, size, d_errorFlag);

        err = hipMemcpy(&h_errorFlag, d_errorFlag, sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess) throw CudaMemcpyExceptionErrorFlag();

        if (h_errorFlag != 0) {
            throw DivisionByZeroException();
        }

        // Perform the division if no zero values are found
        matrixDivideKernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_a, gpu_b, gpu_c, size);

        err = hipGetLastError();
        if (err != hipSuccess) throw KernelLaunchException();

        // Copy result back to host
        err = hipMemcpy(c, gpu_c, size * sizeof(double), hipMemcpyDeviceToHost);
        if (err != hipSuccess) throw CudaMemcpyExceptionC();

    } catch (const CudaMallocExceptionGpuA& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -1;
     } catch (const CudaMallocExceptionGpuB& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -2;
     } catch (const CudaMallocExceptionGpuC& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -3;
     } catch (const CudaMallocExceptionErrorFlag& e){
         CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
         return -4;
     } catch (const CudaMemcpyExceptionA& e){
         CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
         return -5;
     } catch (const CudaMemcpyExceptionB& e){
         CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
         return -6;
     } catch (const CudaMemcpyExceptionC& e){
         CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
         return -7;
     } catch (const KernelLaunchException& e){
         CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
         return -8;
     } catch (const DivisionByZeroException& e){
         CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
         return -9;
     } catch (const std::runtime_error& e) {
         CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
         return -1;
    }

    // Free memory
    CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
    return 0;  // Return success code
}

/*
 * CUDA interface function: Multiply two Matrices
 *
 *      c = a * b, return error_code (-1) or success (0)
 */
extern "C" int matrix_multiply(double* a, double* b, double* c, int rows, int cols) {
    int size = rows * cols;
    double* gpu_a = nullptr;
    double* gpu_b = nullptr;
    double* gpu_c = nullptr;
    int* d_errorFlag = 0;

    try {
        hipError_t err;
        // Allocate device memory
        err = hipMalloc((void**)&gpu_a, size * sizeof(double));
        if (err != hipSuccess) throw CudaMallocExceptionGpuA();

        err = hipMalloc((void**)&gpu_b, size * sizeof(double));
        if (err != hipSuccess) throw CudaMallocExceptionGpuB();

        err = hipMalloc((void**)&gpu_c, size * sizeof(double));
        if (err != hipSuccess) throw CudaMallocExceptionGpuC();

        err = hipMalloc((void**)&d_errorFlag, sizeof(int));
        if (err != hipSuccess) throw CudaMallocExceptionErrorFlag();

        // Copy data to device
        err = hipMemcpy(gpu_a, a, size * sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) throw CudaMemcpyExceptionA();

        err = hipMemcpy(gpu_b, b, size * sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) throw CudaMemcpyExceptionA();

        // Launch kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_a, gpu_b, gpu_c, rows, cols);

        err = hipGetLastError();
        if (err != hipSuccess) throw KernelLaunchException();

        // Copy result back to host
        err = hipMemcpy(c, gpu_c, size * sizeof(double), hipMemcpyDeviceToHost);
        if (err != hipSuccess) throw CudaMemcpyExceptionC();

    } catch (const CudaMallocExceptionGpuA& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -1;
    } catch (const CudaMallocExceptionGpuB& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -2;
    } catch (const CudaMallocExceptionGpuC& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -3;
    } catch (const CudaMallocExceptionErrorFlag& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -4;
    } catch (const CudaMemcpyExceptionA& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -5;
    } catch (const CudaMemcpyExceptionB& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -6;
    } catch (const CudaMemcpyExceptionC& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -7;
    } catch (const KernelLaunchException& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -8;
    } catch (const DivisionByZeroException& e){
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -9;
    } catch (const std::runtime_error& e) {
        CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);
        return -1;
    }
    // Free memory
    CUDA_FREE_ALL(gpu_a,gpu_b,gpu_c);

    return 0;  // Return success code
}