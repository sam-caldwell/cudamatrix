#include <hip/hip_runtime.h>
#include <iostream>
#include "exceptions.h"
#include "printMatrix.h"
#include "freeMatrixMemory.h"
#include "initializeGpuMatrix.h"
#include "initializeGpuErrorFlag.h"
#include "copyGpuMatrixToHost.h"
#include "captureGpuErrors.h"
#include "waitOnKernel.h"
#include "kernels/add.h"
#include "kernels/divide.h"
#include "kernels/multiply.h"

/*
 * CUDA interface function: Add two Matrices
 *
 *      c = a + b, return error_code (-1) or success (0)
 */
extern "C" int matrixAdd(double *matrixA, double *matrixB, double *matrixC, int rows, int cols) {
    const int size = rows * cols;
    double* gpuMatrixA = nullptr;
    double* gpuMatrixB = nullptr;
    double* gpuMatrixC = nullptr;
    int gpuError = 0;
    try{
        initializeGpuErrorFlag(&gpuError);
        initializeGpuMatrix(matrixA, gpuMatrixA, size, true);
        initializeGpuMatrix(matrixB, gpuMatrixB, size, true);
        initializeGpuMatrix(matrixC, gpuMatrixC, size, false);

        std::cout << "setup state:" << std::endl;
        print1dMatrix(gpuMatrixA, size);
        print1dMatrix(gpuMatrixB, size);
        print1dMatrix(gpuMatrixC, size);

        captureGpuErrors(&gpuError);

        std::cout << "launch kernel:" << std::endl;
        dim3 blockSize(16, 16);
        dim3 numBlocks((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);
        matrixAddKernel<<<numBlocks, blockSize>>>(gpuMatrixA, gpuMatrixB, gpuMatrixC, rows, cols, &gpuError);
        hipDeviceSynchronize();

        captureGpuErrors(&gpuError);

        std::cout << "end state:" << std::endl;
        print1dMatrix(gpuMatrixA, size);
        print1dMatrix(gpuMatrixB, size);
        print1dMatrix(gpuMatrixC, size);


    } catch (const CudaException& e){
        freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, &gpuError);
        return e.error();
    } catch (const ProgramError& e){
        freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, &gpuError);
        return e.error();
    } catch (const std::runtime_error& e) {
        const int unhandledException = -65535;
        freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, &gpuError);
        return unhandledException;
    }
    freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, &gpuError);
    return static_cast<int>(hipSuccess);
}

/*
 * CUDA interface function: Divide two Matrices
 *
 *      c = a / b, return error_code (-1) or success (0)
 *
 * BAD: NEEDS WORK STILL
 *
 */
extern "C" int matrixDivide(double* matrixA, double* matrixB, double* matrixC, int rows, int cols) {
    int size = rows * cols;
    double* gpuMatrixA = nullptr;
    double* gpuMatrixB = nullptr;
    double* gpuMatrixC = nullptr;
    int* gpuError = nullptr;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    try {
        initializeGpuMatrix(matrixA, gpuMatrixA, size, true);
        initializeGpuMatrix(matrixB, gpuMatrixB, size, true);
        initializeGpuMatrix(matrixC, gpuMatrixC, size, false);
        initializeGpuErrorFlag(gpuError);

        matrixDivideKernel<<<blocksPerGrid, threadsPerBlock>>>(gpuMatrixA, gpuMatrixB, gpuMatrixC, size, gpuError);
        waitOnKernel();

        captureGpuErrors(gpuError);
        copyGpuMatrixToHost(matrixC, gpuMatrixC, size);
    } catch (const CudaException& e){
        freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, gpuError);
        return e.error();
    } catch (const DivisionByZeroException& e){
        freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, gpuError);
        return e.error();
    } catch (const std::runtime_error& e) {
        const int unhandledException = -65535;
        freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, gpuError);
        return unhandledException;
    }
    freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, gpuError);
    return static_cast<int>(hipSuccess);  // Return success code
}

/*
 * CUDA interface function: Multiply two Matrices
 *
 *      c = a * b, return error_code (-1) or success (0)
 *
 * BAD: NEEDS WORK STILL
 *
 */
extern "C" int matrixMultiply(double* matrixA, double* matrixB, double* matrixC, int rows, int cols) {
    int size = rows * cols;
    double* gpuMatrixA = nullptr;
    double* gpuMatrixB = nullptr;
    double* gpuMatrixC = nullptr;
    int* gpuError = nullptr;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    try {
        initializeGpuMatrix(matrixA, gpuMatrixA, size, true);
        initializeGpuMatrix(matrixB, gpuMatrixB, size, true);
        initializeGpuMatrix(matrixC, gpuMatrixC, size, false);
        initializeGpuErrorFlag(gpuError);

        matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(gpuMatrixA, gpuMatrixB, gpuMatrixC, rows, cols, gpuError);
        waitOnKernel();

        captureGpuErrors(gpuError);
        copyGpuMatrixToHost(matrixC, gpuMatrixC, size);
    } catch (const CudaException& e){
        freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, gpuError);
        return e.error();
    } catch (const std::runtime_error& e) {
        const int unhandledException = -65535;
        freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, gpuError);
        return unhandledException;
    }
    freeMatrixMemory(gpuMatrixA,gpuMatrixB,gpuMatrixC, gpuError);
    return static_cast<int>(hipSuccess);  // Return success code
}
