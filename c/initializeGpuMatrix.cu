
void initializeGpuMatrix(double* HostMatrix, double*& GpuMatrix, int size, bool copyData){
    hipError_t err;
    err = hipMalloc((void**)&GpuMatrix, size * sizeof(double));
    if (err != hipSuccess) throw CudaException(err);

    if (copyData){
        err = hipMemcpy(GpuMatrix, HostMatrix, size * sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) throw CudaException(err);
    }
}