
#include <hip/hip_runtime.h>

void freeMatrixMemory(double* gpuMatrixA, double* gpuMatrixB, double* gpuMatrixC, int *error){

    if (gpuMatrixA) hipFree(gpuMatrixA);

    if (gpuMatrixB) hipFree(gpuMatrixB);

    if (gpuMatrixC) hipFree(gpuMatrixC);

    if (error) hipFree(error);

}
