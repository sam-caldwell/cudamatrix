
void copyGpuMatrixToHost(double* HostMatrix, double* GpuMatrix, int size){

    hipError_t err;

    err = hipMemcpy(HostMatrix, GpuMatrix, size * sizeof(double), hipMemcpyDeviceToHost);

    if (err != hipSuccess) throw CudaException(err);

}