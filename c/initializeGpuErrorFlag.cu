
void initializeGpuErrorFlag(int*& gpuError){

    hipError_t err;

    if (gpuError == nullptr) {

        err = hipMalloc((void**)&gpuError, sizeof(int));

        if (err != hipSuccess) throw CudaException(err);

        // Initialize the error flag to zero
        err = hipMemset(gpuError, 0, sizeof(int));

        if (err != hipSuccess) throw CudaException(err);

    }

}
